
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


const int TILE_DIM =32;
const int BLOCK_ROWS=8;
const int NUM_REPS= 100;


__global__
void copy(float *odata, float *idata){
    int x=blockIdx.x*TILE_DIM+threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int w=gridDim.x*TILE_DIM;

    for(int j=0;j<TILE_DIM;j+=BLOCK_ROWS)
        odata[(y+j)*w+x]=idata[(y+j)*w+x];

}


__global__
void transpose_naive(float *odata, const float* idata){
    int x=blockIdx.x*TILE_DIM+threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int w=gridDim.x*TILE_DIM;

    for(int j=0;j<TILE_DIM;j+=BLOCK_ROWS)
        odata[x*w +(y+j)]=idata[(y+j)*w+x];

}

int main(){
    const int nx=1024;
    const int ny=1024;
    const int mem_size=nx*ny*sizeof(float);

    dim3 dimGrid(nx/TILE_DIM,ny/TILE_DIM,1);
    dim3 dimBlock(TILE_DIM,BLOCK_ROWS,1); 

    float *h_idata= new float[mem_size];
    float *h_odata= new float[mem_size];
    float *result= new float[mem_size];
    float *d_idata,*d_odata;

    //Alloc memory on device
    hipMallocManaged(&d_idata,mem_size);
    hipMallocManaged(&d_odata,mem_size);


    //Computation on host
    for(int j=0;j<ny;j++)
        for(int i=0;i<nx;i++)
            h_idata[j*nx+i]=(float)(i+j*nx);
    
    //For comparison purpose
    for(int j=0;j<ny;j++)
        for(int i=0;i<nx;i++)
            result[i*ny+j]=h_idata[j*nx+i];


    hipMemcpy(d_idata,h_idata,mem_size,hipMemcpyHostToDevice);

    //Transpose Naive 
    hipMemset(d_odata,0 ,mem_size); 
    transpose_naive<<<dimGrid,dimBlock>>>(d_odata,d_idata);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors 
    float maxError = 0.0f;
    for (int i = 0; i < nx*ny; i++)
        maxError = fmax(maxError, fabs(d_odata[i]-result[i]));
    std::cout << "Max error: " << maxError << std::endl;


    //cleanup
    delete[] h_idata;
    delete[] h_odata;
    delete[] result;

    hipFree(d_idata);
    hipFree(d_odata);


    return 0;



}